
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <stdio.h>

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){

  hello<<<2, 2>>>();
  hipDeviceSynchronize();
}
