
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

const int DSIZE = 4096;
const int block_size = 256;

__global__ void vadd(const float *A, const float *B, float *C, int ds){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < ds)
    C[idx] = A[idx]+B[idx];
}

int main(){
  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
  h_A = new float[DSIZE];
  h_B = new float[DSIZE];
  h_C = new float[DSIZE];
  for (int i = 0; i < DSIZE; i++){
    h_A[i] = rand()/(float)RAND_MAX;
    h_B[i] = rand()/(float)RAND_MAX;
    h_C[i] = 0;
  }
  hipMalloc(&d_A, DSIZE*sizeof(float));
  hipMalloc(&d_B, DSIZE*sizeof(float));
  hipMalloc(&d_C, DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  
  hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  
  vadd<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");
  
  hipMemcpy(h_C, d_C, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckErrors("kernel execution failure or hipMemcpy D2H failure");
  
  printf("A[0] = %f\n", h_A[0]);
  printf("B[0] = %f\n", h_B[0]);
  printf("C[0] = %f\n", h_C[0]);
  
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  
  return 0;
}
